#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../commonUtils.cpp"

constexpr int N = 256;
constexpr int SIZE = N*N;
constexpr int BLOCK_SIZE = 16;

__global__ void matMul(const float *a, const float *b, float *c, int N) {
	int x = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int y = threadIdx.y + blockIdx.y * BLOCK_SIZE;

  c[x * N + y] = 0;
  for (int i = 0; i < N; i++) {
    c[x * N + y] += a[x * N + i] * b[i * N + y];
  }
}

__global__ void matMulShared(float* a, float* b, float* c, size_t N) {
	float sum = 0.0;

	unsigned __int16 aBegin = N * BLOCK_SIZE * blockIdx.y;
	unsigned __int16 aEnd = aBegin + N - 1;
  
	unsigned __int16 aStep = BLOCK_SIZE;

	unsigned __int16 bBegin = BLOCK_SIZE * blockIdx.x;
	unsigned __int16 bStep = BLOCK_SIZE * N;

	for (unsigned __int16 ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep) {
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[threadIdx.y][threadIdx.x] = a[ia + N * threadIdx.y + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = b[ib + N * threadIdx.y + threadIdx.x];

		__syncthreads();

		for (unsigned __int16 k = 0; k < BLOCK_SIZE; k++)
		{
			sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
		}

		__syncthreads();
	}

	int ic = N * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
	c[ic + N * threadIdx.y + threadIdx.x] = sum;
}

__host__ void matMulCPU(const float *a, const float *b, float *c, int N) {
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            c[i * N + j] = 0;
            for (int h = 0; h < N; ++h)
            {
                c[i * N + j] += a[i * N + h] * b[h * N + j];
            }
        }
    }
}

void printDeviceInfo();
std::string formatBytes(size_t bytes);
void printMatrix(float *matrix, int N);
bool matrixEquals(float *a, float *b, int N);

int main() {
  printDeviceInfo();

  float *h_A = new float[SIZE];
  float *h_B = new float[SIZE];
  float *h_C = new float[SIZE];
  float *hd_C = new float[SIZE];
  float *hds_C = new float[SIZE];

  for (int i = 0; i < SIZE; i++) {
    h_A[i] = (rand() % 1000) / 1000.0f;
    h_B[i] = (rand() % 1000) / 1000.0f;
  }

  double cpuMills=0;
  {
    clock_t cpuStart, cpuEnd;
    cpuStart = clock();
    matMulCPU(h_A, h_B, h_C, N);
    cpuEnd = clock();
    cpuMills = ((cpuEnd - cpuStart)/CLOCKS_PER_SEC);
  }

  float *d_A, *d_B, *ds_C, *d_C;
  hipMalloc(&d_A, SIZE * sizeof(float));
  hipMalloc(&d_B, SIZE * sizeof(float));
  hipMalloc(&d_C, SIZE * sizeof(float));
  hipMalloc(&ds_C, SIZE * sizeof(float));

  hipMemcpy(d_A, h_A, SIZE * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, SIZE * sizeof(float), hipMemcpyHostToDevice);

	constexpr int blockCount = ((N + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 blockConf(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridConf(blockCount, blockCount);

  float cudaMills = 0;
  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matMul<<<gridConf, blockConf>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipMemcpy(hd_C, d_C, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudaMills, start, stop);
  }

  float cudaMillsShared = 0;
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulShared), hipFuncCachePreferShared);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matMulShared<<<gridConf, blockConf>>>(d_A, d_B, ds_C, N);
    hipEventRecord(stop);

    hipMemcpy(hds_C, ds_C, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudaMillsShared, start, stop);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  std::cout << "Matrix equals: hk " << matrixEquals(h_C, hd_C, N) << "; hks " << matrixEquals(h_C, hds_C, N) << std::endl;
  std::cout << "Kernel execution time: " << cudaMills << " ms" << std::endl;
  std::cout << "Kernel shared mem execution time: " << cudaMillsShared << " ms" << std::endl;
  std::cout << "CPU execution time: " << cpuMills << " ms" << std::endl;

  return 0;
}

void printDeviceInfo() {
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  std::cout << "Device Name: " << devProp.name << std::endl;
  std::cout << "Total Global Memory: " << formatBytes(devProp.totalGlobalMem) << std::endl;
  std::cout << "Shared Memory per Block: " << formatBytes(devProp.sharedMemPerBlock) << std::endl;
  std::cout << "Registers per Block: " << devProp.regsPerBlock << std::endl;
  std::cout << "Total Constant Memory: " << formatBytes(devProp.totalConstMem) << std::endl;
  std::cout << "L2 Cache Size (bytes): " << formatBytes(devProp.l2CacheSize) << std::endl;
}

std::string formatBytes(size_t bytes) {
  std::string suffix = "b";
  double value = bytes;
  if (bytes >= 1024) {
    suffix = "KB";
    value = bytes / 1024.0;
  }
  if (bytes >= 1024 * 1024) {
    suffix = "MB";
    value = bytes / (1024.0 * 1024.0);
  }
  if (bytes >= 1024 * 1024 * 1024) {
    suffix = "GB";
    value = bytes / (1024.0 * 1024.0 * 1024.0);
  }
  return std::to_string(value) + suffix;
}

void printMatrix(float *matrix, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      std::cout << matrix[i * N + j] << " ";
    }
    std::cout << std::endl;
  }
}

bool matrixEquals(float *a, float *b, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      if (fabs(a[i * N + j] - b[i * N + j]) >= 0.01f) {
        std::cout << "a[" << i << "][" << j << "] = " << a[i * N + j] << " != b[" << i << "][" << j << "] = " << b[i * N + j] << std::endl;
        return false;
      }
    }
  }
  return true;
}